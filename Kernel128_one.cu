#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <assert.h>
#include <xmmintrin.h>
#include <immintrin.h>

#include "hipDNN.h"
#include "util.h"
#include "Kernel128_one.h"


#define cudaCheckError() {																\
	hipError_t e=hipGetLastError();													\
	if(e!=hipSuccess) {																\
		printf("Cuda failure %s:%d:'%s'\n",__FILE__,__LINE__,hipGetErrorString(e));	\
		exit(EXIT_FAILURE);																\
	}																					\
}

__global__ void kernel_512_one_128(float *A, float *B, float *bnBias, float *bnScale, float *C) {
	int tile = blockIdx.x, in_channel = threadIdx.x, line = threadIdx.y;
	int ind = line*128 + in_channel;

	extern __shared__ float shared_[];
	float *weights = shared_ + 512*4, *output = weights + 128*64, *input = shared_;
	float *bias = output + 4*128, *scale = bias + 128;

	for (int i = 0; i < 4; i++)
		input[ind + i*512] = A[tile*2048 + i*512 + ind];
	bias[in_channel] = bnBias[in_channel];
	scale[in_channel] = bnScale[in_channel];
	output[ind] = 0.0f;
	__syncthreads();

	for (int k = 0; k < 512; k += 64) {
		float *B_start = B + k*128;
		for (int i = 0; i < 16; i++)
			weights[ind + i*512] = B_start[i*512 + ind];
		__syncthreads();

		float *A_start = input + k;
		for (int p = 0; p < 64; p++) {
			output[ind] += A_start[line*512 + p] * weights[in_channel + p*128];
		}
		__syncthreads();
	}

	float *C_start = C + tile*512, res = scale[in_channel] * output[ind] + bias[in_channel];
	C_start[ind] = res > 0 ? res : 0;
}


int kernel_128_1_in() {
	float *input = get_parameter(inputName128one, 14*14*512);
	float *weight = get_parameter(weightName128one, 128*512);

	float *bnBias = get_parameter(bnBiasName128one, 128);
	float *bnScale = get_parameter(bnScaleName128one, 128);
	float *bnBias_myKernel = get_parameter(bnBias_myKernel_Name128one, 128);
	float *bnScale_myKernel = get_parameter(bnScale_myKernel_Name128one, 128);
	float *eMeanName = get_parameter(eMeanName128one, 128);
	float *eVarName = get_parameter(eVarName128one, 128);

	float *input_, *output_, *weight_, *bnBias_, *bnScale_, *eMeanName_, *eVarName_;

	int nInput = 14*14*512, nOutput = 14*14*128, nWeights = 128*512;
	float tmp[nOutput], tmp_cudnn[nOutput];

	uint64_t nT1 = 0, nT2 = 0, nT1_cudnn = 0, nT2_cudnn = 0;
	hipError_t s;

	/////////////////////////////////

	// My Kernel

	/////////////////////////////////

	/*  1. Data preparation  */
	hipMalloc((void **) &input_, nInput<<3);
	hipMalloc((void **) &output_, nOutput<<2);
	hipMalloc((void **) &weight_, nWeights<<2);
	hipMalloc((void **) &bnBias_, 128<<2);
	hipMalloc((void **) &bnScale_, 128<<2);

	hipMemcpy(input_, input, nInput<<2, hipMemcpyHostToDevice);
	hipMemcpy(weight_, weight, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnBias_, bnBias_myKernel, 128<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnScale_, bnScale_myKernel, 128<<2, hipMemcpyHostToDevice);


	/*  2. Computing  */
	nT1 = getTimeMicroseconds64();

	kernel_512_one_128 <<<dim3(49), dim3(128, 4), (4*512 + 64*128 + 4*128 + 2*128)<<2 >>> (input_, weight_, bnBias_, bnScale_, output_);

	//cudaCheckError();
	hipDeviceSynchronize();

	nT2 = getTimeMicroseconds64();
	printf("TotalTime = %d us\n", nT2-nT1);


	/*  3. Copy back and free  */
	s = hipMemcpy(tmp, output_, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	cudaCheckError();

	free(bnBias_myKernel);
	free(bnScale_myKernel);


	/////////////////////////////////

	// cuDNN

	/////////////////////////////////

	/*  1. Data preparation  */
	hipMalloc((void **) &eMeanName_, 128<<2);
	hipMalloc((void **) &eVarName_, 128<<2);

	hipMemcpy(bnBias_, bnBias, 128<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnScale_, bnScale, 128<<2, hipMemcpyHostToDevice);
	hipMemcpy(eMeanName_, eMeanName, 128<<2, hipMemcpyHostToDevice);
	hipMemcpy(eVarName_, eVarName, 128<<2, hipMemcpyHostToDevice);

	weight = transpose(weight, 128, 512);
	hipMemcpy(weight_, weight, nWeights<<2, hipMemcpyHostToDevice);

	/*  2. cuDNN preparation  */
	hipdnnStatus_t status;
	float one = 1.0, zero = 0.0;
	int size;

	hipdnnHandle_t handle;
	status = hipdnnCreate(&handle);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed1\n");

	hipdnnTensorDescriptor_t xdesc, ydesc;
	hipdnnFilterDescriptor_t wdesc; // HIPDNN_TENSOR_NHWC, HIPDNN_TENSOR_NCHW
	status = hipdnnCreateTensorDescriptor(&xdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed2\n");
	status = hipdnnSetTensor4dDescriptor(xdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 512, 14, 14);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed3\n");
	status = hipdnnCreateTensorDescriptor(&ydesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed4\n");
	status = hipdnnSetTensor4dDescriptor(ydesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 14, 14);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5\n");
	status = hipdnnCreateFilterDescriptor(&wdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed6\n");
	status = hipdnnSetFilter4dDescriptor(wdesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 128, 512, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed7\n");

	hipdnnConvolutionDescriptor_t conv_desc;
	status = hipdnnCreateConvolutionDescriptor(&conv_desc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed10\n");
	status = hipdnnSetConvolution2dDescriptor(conv_desc, 0,0, 1,1,1,1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT); //HIPDNN_CONVOLUTION
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed11\n");

	hipdnnActivationDescriptor_t act_desc;
	status = hipdnnCreateActivationDescriptor(&act_desc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed12\n");
	status = hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed13\n");

	hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc;
	status = hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVarDesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed14\n");
	status = hipdnnSetTensor4dDescriptor(bnScaleBiasMeanVarDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 128, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed15\n");

	hipdnnConvolutionFwdAlgo_t algo = (hipdnnConvolutionFwdAlgo_t)0;

	status = hipdnnGetConvolutionForwardWorkspaceSize(handle,
	   xdesc,
	   wdesc,
	   conv_desc,
	   ydesc,
	   algo,
	   (size_t *)&(size));
	float *extra;
	hipMalloc((void **) &extra, size);


	/*  3. Computing  */
	nT1_cudnn = getTimeMicroseconds64();

	status = hipdnnConvolutionForward(handle, &one,
		xdesc, input_, wdesc, weight_,
		conv_desc, algo,
		extra, size, &zero,
		ydesc, output_);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed1\n");

	status = hipdnnBatchNormalizationForwardInference(handle, HIPDNN_BATCHNORM_SPATIAL,
		&one, &zero,
		ydesc, output_, ydesc, output_,
		bnScaleBiasMeanVarDesc, bnScale_, bnBias_, eMeanName_, eVarName_, HIPDNN_BN_MIN_EPSILON);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed2\n");

	status = hipdnnActivationForward(handle, act_desc, &one,
		ydesc, output_, &zero,
		ydesc, output_);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed3\n");

	hipDeviceSynchronize();
	nT2_cudnn = getTimeMicroseconds64();
	printf("cuDNN TotalTime = %d us\n", nT2_cudnn-nT1_cudnn);


	/*  4. Copy back and free  */
	s = hipMemcpy(tmp_cudnn, output_, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));

	hipFree(extra);
	hipFree(input_);
	hipFree(output_);
	hipFree(weight_);

	hipFree(bnScale_);
	hipFree(bnBias_);
	hipFree(eMeanName_);
	hipFree(eVarName_);

	free(input);
	free(weight);

	free(bnScale);
	free(bnBias);
	free(eMeanName);
	free(eVarName);

	output_checker(tmp, tmp_cudnn, 14, 128, 0);

	return ((nT2-nT1) << 16) | (nT2_cudnn-nT1_cudnn);
}



__global__ void kernel_128_one_512(float *A, float *B, float *bnBias, float *bnScale, float *C) {
	int tile = blockIdx.x, part = blockIdx.y, in_channel = threadIdx.x, line = threadIdx.y;
	int ind = line*128 + in_channel;

	extern __shared__ float shared_[];
	float *weights = shared_ + 128*4, *output = weights + 128*64, *input = shared_;
	float *bias = output + 4*128, *scale = bias + 128;

	input[ind] = A[tile * 512 + ind];
	bias[in_channel] = bnBias[part*128 + in_channel];
	scale[in_channel] = bnScale[part*128+ in_channel];
	output[ind] = 0.0f;
	__syncthreads();

	for (int k = 0; k < 128; k += 64) {
		for (int i = 0; i < 16; i++)
			weights[ind + 512*i] = B[(k + i*4 + line)*512 + part*128 + in_channel];
		__syncthreads();

		float *A_start = input + k;
		for (int p = 0; p < 64; p++) {
			output[ind] += A_start[line*128 + p] * weights[in_channel + p*128];
		}
		__syncthreads();
	}

	float *C_start = C + tile*2048 + part*128;
	float res = scale[in_channel] * output[ind] + bias[in_channel];
	C_start[line * 512 + in_channel] = res;
}


int kernel_128_1_out() {
	float *input = get_parameter(inputName128one, 14*14*128);
	float *weight = get_parameter(weightName128one, 128*512);

	float *bnBias = get_parameter(bnBiasName128one, 512);
	float *bnScale = get_parameter(bnScaleName128one, 512);
	float *bnBias_myKernel = get_parameter(bnBias_myKernel_Name128one, 512);
	float *bnScale_myKernel = get_parameter(bnScale_myKernel_Name128one, 512);
	float *eMeanName = get_parameter(eMeanName128one, 512);
	float *eVarName = get_parameter(eVarName128one, 512);

	float *input_, *output_, *weight_, *bnBias_, *bnScale_, *eMeanName_, *eVarName_;

	int nInput = 14*14*128, nOutput = 14*14*512, nWeights = 128*512;
	float tmp[nOutput], tmp_cudnn[nOutput];

	uint64_t nT1 = 0, nT2 = 0, nT1_cudnn = 0, nT2_cudnn = 0;
	hipError_t s;

	/////////////////////////////////

	// My Kernel

	/////////////////////////////////

	/*  1. Data preparation  */
	hipMalloc((void **) &input_, nInput<<3);
	hipMalloc((void **) &output_, nOutput<<2);
	hipMalloc((void **) &weight_, nWeights<<2);
	hipMalloc((void **) &bnBias_, 512<<2);
	hipMalloc((void **) &bnScale_, 512<<2);

	hipMemcpy(input_, input, nInput<<2, hipMemcpyHostToDevice);
	hipMemcpy(weight_, weight, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnBias_, bnBias_myKernel, 512<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnScale_, bnScale_myKernel, 512<<2, hipMemcpyHostToDevice);

	/*  2. Computing  */
	nT1 = getTimeMicroseconds64();

	kernel_128_one_512 <<<dim3(49, 4), dim3(128, 4), (4*128 + 64*128 + 4*128 + 2*128)<<2 >>> (input_, weight_, bnBias_, bnScale_, output_);

	//cudaCheckError();
	hipDeviceSynchronize();

	nT2 = getTimeMicroseconds64();
	printf("TotalTime = %d us\n", nT2-nT1);


	/*  3. Copy back and free  */
	s = hipMemcpy(tmp, output_, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	cudaCheckError();

	free(bnBias_myKernel);
	free(bnScale_myKernel);


	/////////////////////////////////

	// cuDNN

	/////////////////////////////////

	/*  1. Data preparation  */
	hipMalloc((void **) &eMeanName_, 512<<2);
	hipMalloc((void **) &eVarName_, 512<<2);

	hipMemcpy(bnBias_, bnBias, 512<<2, hipMemcpyHostToDevice);
	hipMemcpy(bnScale_, bnScale, 512<<2, hipMemcpyHostToDevice);
	hipMemcpy(eMeanName_, eMeanName, 512<<2, hipMemcpyHostToDevice);
	hipMemcpy(eVarName_, eVarName, 512<<2, hipMemcpyHostToDevice);

	weight = transpose(weight, 512, 128);
	hipMemcpy(weight_, weight, nWeights<<2, hipMemcpyHostToDevice);

	/*  2. cuDNN preparation  */
	hipdnnStatus_t status;
	float one = 1.0, zero = 0.0;
	int size;

	hipdnnHandle_t handle;
	status = hipdnnCreate(&handle);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed1\n");

	hipdnnTensorDescriptor_t xdesc, ydesc;
	hipdnnFilterDescriptor_t wdesc; // HIPDNN_TENSOR_NHWC, HIPDNN_TENSOR_NCHW
	status = hipdnnCreateTensorDescriptor(&xdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed2\n");
	status = hipdnnSetTensor4dDescriptor(xdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 14, 14);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed3\n");
	status = hipdnnCreateTensorDescriptor(&ydesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed4\n");
	status = hipdnnSetTensor4dDescriptor(ydesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 512, 14, 14);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5\n");
	status = hipdnnCreateFilterDescriptor(&wdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed6\n");
	status = hipdnnSetFilter4dDescriptor(wdesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 512, 128, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed7\n");

	hipdnnConvolutionDescriptor_t conv_desc;
	status = hipdnnCreateConvolutionDescriptor(&conv_desc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed10\n");
	status = hipdnnSetConvolution2dDescriptor(conv_desc, 0,0, 1,1,1,1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT); //HIPDNN_CONVOLUTION
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed11\n");

	hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc;
	status = hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVarDesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed14\n");
	status = hipdnnSetTensor4dDescriptor(bnScaleBiasMeanVarDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 512, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed15\n");

	hipdnnConvolutionFwdAlgo_t algo = (hipdnnConvolutionFwdAlgo_t)0;

	status = hipdnnGetConvolutionForwardWorkspaceSize(handle,
	   xdesc,
	   wdesc,
	   conv_desc,
	   ydesc,
	   algo,
	   (size_t *)&(size));
	float *extra;
	hipMalloc((void **) &extra, size);


	/*  3. Computing  */
	nT1_cudnn = getTimeMicroseconds64();

	status = hipdnnConvolutionForward(handle, &one,
		xdesc, input_, wdesc, weight_,
		conv_desc, algo,
		extra, size, &zero,
		ydesc, output_);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed1\n");

	status = hipdnnBatchNormalizationForwardInference(handle, HIPDNN_BATCHNORM_SPATIAL,
		&one, &zero,
		ydesc, output_, ydesc, output_,
		bnScaleBiasMeanVarDesc, bnScale_, bnBias_, eMeanName_, eVarName_, HIPDNN_BN_MIN_EPSILON);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed2\n");

	hipDeviceSynchronize();
	nT2_cudnn = getTimeMicroseconds64();
	printf("cuDNN TotalTime = %d us\n", nT2_cudnn-nT1_cudnn);


	/*  4. Copy back and free  */
	s = hipMemcpy(tmp_cudnn, output_, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));

	hipFree(extra);
	hipFree(input_);
	hipFree(output_);
	hipFree(weight_);

	hipFree(bnScale_);
	hipFree(bnBias_);
	hipFree(eMeanName_);
	hipFree(eVarName_);

	free(input);
	free(weight);

	free(bnScale);
	free(bnBias);
	free(eMeanName);
	free(eVarName);

	output_checker(tmp, tmp_cudnn, 14, 512, 0);

	return ((nT2-nT1) << 16) | (nT2_cudnn-nT1_cudnn);
}
